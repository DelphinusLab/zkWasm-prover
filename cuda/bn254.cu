#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include "bn254.cuh"

__global__ void _eval_lookup_z_step1(
    Bn254FrField *z,
    Bn254FrField *permuted_input,
    Bn254FrField *permuted_table,
    Bn254FrField *beta_gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    z[i] = (permuted_input[i] + beta_gamma[0]) * (permuted_table[i] + beta_gamma[1]);
}

__global__ void _eval_lookup_z_batch_invert(
    Bn254FrField *z,
    Bn254FrField *tmp,
    int size_per_worker)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Bn254FrField t(1);
    Bn254FrField u(1);
    for (int j = i * size_per_worker; j < i * size_per_worker + size_per_worker; j++) {
        u = t * z[j];
        tmp[j] = t;
        t = u;
    }

    t = t.inv();

    for (int j = i * size_per_worker + size_per_worker - 1; j >= i * size_per_worker; j--) {
        u = z[j];
        z[j] = t * tmp[j];
        t = t * u;
        tmp[j] = u;
    }
}

__global__ void _eval_lookup_z_step2(
    Bn254FrField *z,
    Bn254FrField *input,
    Bn254FrField *table,
    Bn254FrField *beta_gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    z[i] = z[i] * (input[i] + beta_gamma[0]) * (table[i] + beta_gamma[1]);
}

__global__ void _eval_lookup_z_product_batch(
    Bn254FrField *z,
    Bn254FrField *res,
    int size_per_worker)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i != 0) {
        i--;
        Bn254FrField t(1);
        for (int j = i * size_per_worker; j < i * size_per_worker + size_per_worker; j++) {
            t *= z[j];
        }
        res[i + 1] = t;
    } else {
        res[i] = Bn254FrField(1);
    }
}

__global__ void _eval_lookup_z_product_single_spread(
    Bn254FrField *res,
    int size_per_worker)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 1; i < size_per_worker; i++) {
        res[i] *= res[i - 1];
    }
}

__global__ void _eval_lookup_z_product_batch_spread(
    Bn254FrField *z,
    Bn254FrField *res,
    int size_per_worker)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    z[i * size_per_worker] *= res[i];
    for (int j = i * size_per_worker + 1; j < i * size_per_worker + size_per_worker; j++) {
        z[j] *= z[j - 1];
    }
}

// Place a Bn254FrField::one() in the front
__global__ void _eval_lookup_z_product_batch_spread_skip(
    Bn254FrField *z,
    Bn254FrField *res,
    int size_per_worker)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Bn254FrField t = res[i];
    Bn254FrField u;
    for (int j = i * size_per_worker; j < i * size_per_worker + size_per_worker; j++) {
        u = z[j] * t;
        z[j] = t;
        t = u;
    }
}

__global__ void _poly_eval(
    Bn254FrField *p,
    Bn254FrField *out,
    const Bn254FrField *x,
    int deg)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = p[i * 2] + p[i * 2 + 1] * x[deg];
}

__global__ void _msm_mont_unmont(
    Bn254G1Affine *p,
    Bn254FrField *s,
    bool mont,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    for (int i = start; i < end; i++)
    {
        if (mont)
        {
            s[i].mont_assign();
        }
        else
        {
            s[i].unmont_assign();
        }
    }
}

__global__ void _msm_core(
    Bn254G1 *res,
    const Bn254G1Affine *p,
    Bn254FrField *s,
    int n)
{
    int group_idx = blockIdx.x;
    int worker = blockDim.x * gridDim.y;
    int size_per_worker = (n + worker - 1) / worker;
    int inner_idx = threadIdx.x;
    int window_idx = inner_idx + blockIdx.y * blockDim.x;
    int start = window_idx * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    __shared__ Bn254G1 thread_res[128];

    Bn254G1 buckets[256];

    for (int i = start; i < end; i++)
    {
        int v = s[i].get_8bits(group_idx);
        if (v--)
        {
            buckets[v] = buckets[v] + p[i];
        }
    }

    if (end > start)
    {
        Bn254G1 round;
        Bn254G1 acc;
        for (int i = 254; i >= 0; i--)
        {
            round = round + buckets[i];
            acc = acc + round;
        }

        thread_res[inner_idx] = acc;
    }

    __syncthreads();
    if (inner_idx == 0)
    {
        Bn254G1 acc;
        for (int i = 0; i < blockDim.x; i++)
        {
            acc = acc + thread_res[i];
        }
        res[group_idx + blockIdx.y * gridDim.x] = acc;
    }
}

__device__ uint bit_reverse(uint n, uint bits)
{
    uint r = 0;
    for (int i = 0; i < bits; i++)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}

__device__ Bn254FrField pow_lookup(const Bn254FrField *bases, uint exponent)
{
    Bn254FrField res(1);
    uint i = 0;
    while (exponent > 0)
    {
        if (exponent & 1)
            res = res * bases[i];
        exponent = exponent >> 1;
        i++;
    }
    return res;
}

// Learn from ec-gpu
__global__ void _ntt_core(
    const Bn254FrField *_x,
    Bn254FrField *_y,
    const Bn254FrField *pq,
    const Bn254FrField *omegas,
    uint n,     // Number of elements
    uint log_p, // Log2 of `p` (Read more in the link above)
    uint deg,   // 1=>radix2, 2=>radix4, 3=>radix8, ...
    uint max_deg,
    uint grids) // Maximum degree supported, according to `pq` and `omegas`
{
    uint lid = threadIdx.x;
    uint lsize = blockDim.x;
    uint t = n >> deg;
    uint p = 1 << log_p;

    uint count = 1 << deg;
    uint counth = count >> 1;
    uint counts = count / lsize * lid;
    uint counte = counts + count / lsize;

    const uint pqshift = max_deg - deg;

    for (uint gridIdx = 0; gridIdx < grids; gridIdx++)
    {
        uint index = blockIdx.x + gridIdx * gridDim.x;
        uint k = index & (p - 1);

        const Bn254FrField *x = _x + index;
        Bn254FrField *y = _y + ((index - k) << deg) + k;

        __shared__ Bn254FrField u[512];
        uint base_exp = (n >> log_p >> deg) * k;
        for (uint i = counts; i < counte; i++)
        {
            u[i] = omegas[base_exp * i] * x[i * t];
        }
        __syncthreads();

        for (uint rnd = 0; rnd < deg; rnd++)
        {
            const uint bit = counth >> rnd;
            for (uint i = counts >> 1; i < counte >> 1; i++)
            {
                const uint di = i & (bit - 1);
                const uint i0 = (i << 1) - di;
                const uint i1 = i0 + bit;
                Bn254FrField tmp = u[i0];
                u[i0] += u[i1];
                u[i1] = tmp - u[i1];

                if (di != 0)
                    u[i1] = pq[di << rnd << pqshift] * u[i1];
            }

            __syncthreads();
        }

        for (uint i = counts >> 1; i < counte >> 1; i++)
        {
            y[i * p] = u[bit_reverse(i, deg)];
            y[(i + counth) * p] = u[bit_reverse(i + counth, deg)];
        }
    }
}

__global__ void _field_sum(
    Bn254FrField *res,
    Bn254FrField **v,
    Bn254FrField **v_c,
    int *v_rot,
    Bn254FrField *omegas,
    int v_n,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    for (int i = start; i < end; i++)
    {
        Bn254FrField fl(0), fr;
        for (int j = 0; j < v_n; j++)
        {
            int v_i = i;

            int omega_exp = ((n + v_rot[j]) * i) & (n - 1);

            fr = v[j][v_i] * omegas[omega_exp];

            if (v_c[j])
            {
                fr = fr * *v_c[j];
            }

            if (j == 0)
            {
                fl = fr;
            }
            else
            {
                fl += fr;
            }
        }

        res[i] = fl;
    }
}

__global__ void _field_op_batch_mul_sum(
    Bn254FrField *res,
    Bn254FrField **v, // coeff0, a00, a01, null, coeff1, a10, a11, null,
    int *rot,
    int n_v,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;

    Bn254FrField fl(0), fr;
    int v_idx = 0;
    int rot_idx = 0;
    while (v_idx < n_v)
    {
        fr = *v[v_idx++]; // first one is coeff
        while (v[v_idx])
        {
            int idx;
            idx = (n + i + rot[rot_idx]) & (n - 1);
            fr = fr * v[v_idx][idx];
            v_idx++;
            rot_idx++;
        }

        fl += fr;
        v_idx++;
    }

    res[i] += fl;
}

__global__ void _field_mul_unaligned(
    Bn254FrField *l,
    Bn254FrField *r,
    int r_n,
    int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    l[i] = l[i] * r[i % r_n];
}

__global__ void _field_op(
    Bn254FrField *res,
    Bn254FrField *l,
    int l_rot,
    Bn254FrField *l_c,
    Bn254FrField *r,
    int r_rot,
    Bn254FrField *r_c,
    int n,
    int op)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Bn254FrField fl, fr;

    if (l)
        if (l_c)
            fl = l[(i + l_rot) % n] * l_c[0];
        else
            fl = l[(i + l_rot) % n];
    else
        fl = l_c[0];

    if (r)
        if (r_c)
            fr = r[(i + r_rot) % n] * r_c[0];
        else
            fr = r[(i + r_rot) % n];
    else
        if (r_c)
            fr = r_c[0];

    // add
    if (op == 0)
    {
        res[i] = fl + fr;
    }
    // mul
    else if (op == 1)
    {
        res[i] = fl * fr;
    }
    // uop
    else if (op == 2)
    {
        res[i] = fl;
    }
    // sub
    else if (op == 3)
    {
        res[i] = fl - fr;
    }
    else
    {
        assert(0);
    }
}

__global__ void _extended_prepare(
    Bn254FrField *s,
    Bn254FrField *coset_powers,
    uint coset_powers_n,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    for (int i = start; i < end; i++)
    {
        int index = i % coset_powers_n;
        if (index != 0)
        {
            s[i] = s[i] * coset_powers[index - 1];
        }
    }
}

__global__ void _permutation_eval_h_p1(
    Bn254FrField *res,
    const Bn254FrField *first_set,
    const Bn254FrField *last_set,
    const Bn254FrField *l0,
    const Bn254FrField *l_last,
    const Bn254FrField *y,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    Bn254FrField t1, t2;

    for (int i = start; i < end; i++)
    {
        t1 = res[i];

        // l_0(X) * (1 - z_0(X)) = 0
        t1 = t1 * y[0];
        t2 = Bn254FrField(1);
        t2 -= first_set[i];
        t2 = t2 * l0[i];
        t1 += t2;

        // l_last(X) * (z_l(X)^2 - z_l(X)) = 0
        t1 = t1 * y[0];
        t2 = last_set[i].sqr();
        t2 -= last_set[i];
        t2 = t2 * l_last[i];
        t1 += t2;

        res[i] = t1;
    }
}

__global__ void _permutation_eval_h_p2(
    Bn254FrField *res,
    const Bn254FrField **set,
    const Bn254FrField *l0,
    const Bn254FrField *l_last,
    const Bn254FrField *y,
    int n_set,
    int rot,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    Bn254FrField t1, t2;

    for (int i = start; i < end; i++)
    {
        int r_prev = (i + n + rot) & (n - 1);
        t1 = res[i];

        for (int j = 1; j < n_set; j++)
        {
            // l_0(X) * (z_i(X) - z_{i-1}(\omega^(last) X)) = 0
            t1 = t1 * y[0];
            t2 = set[j][i] - set[j - 1][r_prev];
            t2 = t2 * l0[i];
            t1 += t2;
        }

        res[i] = t1;
    }
}

__global__ void _permutation_eval_h_l(
    Bn254FrField *res,
    const Bn254FrField *beta,
    const Bn254FrField *gamma,
    const Bn254FrField *p,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    for (int i = start; i < end; i++)
    {
        Bn254FrField t = p[i];
        t = t * beta[0];
        if (i == 0)
        {
            t += gamma[0];
        }
        res[i] += t;
    }
}

__global__ void _permutation_eval_h_r(
    Bn254FrField *res,
    const Bn254FrField *delta,
    const Bn254FrField *gamma,
    const Bn254FrField *value,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int worker = blockDim.x * gridDim.x;
    int size_per_worker = (n + worker - 1) / worker;
    int start = gid * size_per_worker;
    int end = start + size_per_worker;
    end = end > n ? n : end;

    for (int i = start; i < end; i++)
    {
        Bn254FrField t = value[i];
        if (i == 0)
        {
            t += gamma[0];
        }

        if (i == 1)
        {
            t += delta[0];
        }

        res[i] = t;
    }
}

__global__ void _lookup_eval_h(
    Bn254FrField *res,
    const Bn254FrField *input,
    const Bn254FrField *table,
    const Bn254FrField *permuted_input,
    const Bn254FrField *permuted_table,
    const Bn254FrField *z,
    const Bn254FrField *l0,
    const Bn254FrField *l_last,
    const Bn254FrField *l_active_row,
    const Bn254FrField *y,
    const Bn254FrField *beta,
    const Bn254FrField *gamma,
    int rot,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;
    int r_next = (i + rot) & (n - 1);
    int r_prev = (i + n - rot) & (n - 1);

    Bn254FrField t, u, p;
    t = res[i];

    // l_0(X) * (1 - z(X)) = 0
    t = t * *y;
    u = Bn254FrField(1) - z[i];
    u = l0[i] * u;
    t += u;

    // l_last(X) * (z(X)^2 - z(X)) = 0
    t = t * *y;
    u = z[i] * z[i];
    u -= z[i];
    u = l_last[i] * u;
    t += u;

    // (1 - (l_last(X) + l_blind(X))) * (
    //   z(\omega X) (a'(X) + \beta) (s'(X) + \gamma)
    //   - z(X) (\theta^{m-1} a_0(X) + ... + a_{m-1}(X) + \beta)
    //          (\theta^{m-1} s_0(X) + ... + s_{m-1}(X) + \gamma)
    // ) = 0
    t = t * *y;
    u = permuted_input[i] + *beta;
    p = permuted_table[i] + *gamma;
    u = u * p;
    u = u * z[r_next];
    Bn254FrField x = input[i] * table[i];
    u -= z[i] * x;
    u = u * l_active_row[i];
    t += u;

    // l_0(X) * (a'(X) - s'(X)) = 0
    t = t * *y;
    p = permuted_input[i] - permuted_table[i];
    u = l0[i] * p;
    t += u;

    // (1 - (l_last + l_blind)) * (a′(X) − s′(X))⋅(a′(X) − a′(\omega^{-1} X)) = 0
    t = t * *y;
    u = permuted_input[i] - permuted_input[r_prev];
    u = u * p;
    u = u * l_active_row[i];
    t += u;

    res[i] = t;
}


__global__ void _shuffle_eval_h(
    Bn254FrField *res,
    const Bn254FrField *input,
    const Bn254FrField *table,
    const Bn254FrField *z,
    const Bn254FrField *l0,
    const Bn254FrField *l_last,
    const Bn254FrField *l_active_row,
    const Bn254FrField *y,
    int rot,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;
    int r_next = (i + rot) & (n - 1);
    int r_prev = (i + n - rot) & (n - 1);

    Bn254FrField t, u, p;
    t = res[i];

    // l_0(X) * (1 - z(X)) = 0
    t = t * *y;
    u = Bn254FrField(1) - z[i];
    u = l0[i] * u;
    t += u;

    // l_last(X) * (z(X)^2 - z(X)) = 0
    t = t * *y;
    u = z[i] * z[i];
    u -= z[i];
    u = l_last[i] * u;
    t += u;

    // (1 - (l_last(X) + l_blind(X))) *
    // (z(\omega X) (s(X) + \beta^i)- z(X) (a(X) + \beta^i))=0
    t = t * *y;
    u = table[i] * z[r_next];
    u -= input[i] * z[i];
    u = u * l_active_row[i];
    t += u;

    res[i] = t;
}


__global__ void _expand_omega_buffer(
    Bn254FrField *buf,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int workers = gridDim.x * blockDim.x;
    int tasks = n / workers;
    int start = gid * tasks;
    int end = start + tasks;

    start = start < 2 ? 2 : start;
    end = end > n ? n : end;

    Bn254FrField x = buf[1];
    Bn254FrField curr = Bn254FrField::pow(&x, start);

    for (int i = start; i < end; i++)
    {
        buf[i] = curr;
        curr = curr * x;
    }
}

__global__ void _field_mul_zip(
    Bn254FrField *buf,
    Bn254FrField *coeff,
    int coeff_n,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;

    buf[i] = buf[i] * coeff[i % coeff_n];
}

__global__ void _shplonk_h_x_merge(
    Bn254FrField *res,
    Bn254FrField *v,
    Bn254FrField *values,
    Bn254FrField *omegas,
    Bn254FrField *diff_points,
    int diff_points_n,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;

    Bn254FrField t = values[i];
    for (int j = 0; j < diff_points_n; j++) {
        t = t * (omegas[i] - diff_points[j]);
    }
    res[i] = res[i] * v[0] + t;
}

__global__ void _shplonk_h_x_div_points(
    Bn254FrField *values,
    Bn254FrField *omegas,
    Bn254FrField *points,
    int points_n,
    int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = gid;

    Bn254FrField t = omegas[i] - points[0];
    for (int j = 1; j < points_n; j++) {
        t = t * (omegas[i] - points[j]);
    }
    assert(!(t.inv() == Bn254FrField(0)));
    values[i] = values[i] * t.inv();
}

extern "C"
{
    hipError_t field_sum(
        Bn254FrField *res,
        Bn254FrField **v,
        Bn254FrField **v_c,
        int *v_rot,
        Bn254FrField *omegas,
        int v_n,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _field_sum<<<blocks, threads>>>(res, v, v_c, v_rot, omegas, v_n, n);
        return hipGetLastError();
    }

    hipError_t extended_prepare(
        Bn254FrField *s,
        Bn254FrField *coset_powers,
        uint coset_powers_n,
        int size,
        int extended_size,
        int to_coset,
        ihipStream_t *stream)
    {
        int threads = size >= 64 ? 64 : 1;
        int blocks = size / threads;
        if (to_coset)
        {
            _extended_prepare<<<blocks, threads, 0, stream>>>(s, coset_powers, coset_powers_n, extended_size);
        }
        else
        {
            hipMemsetAsync(&s[size], 0, (extended_size - size) * sizeof(Bn254FrField), stream);
            _extended_prepare<<<blocks, threads, 0, stream>>>(s, coset_powers, coset_powers_n, size);
        }
        return hipGetLastError();
    }

    hipError_t field_op_batch_mul_sum(
        Bn254FrField *res,
        Bn254FrField **v, // coeff0, a00, a01, null, coeff1, a10, a11, null,
        int *rot,
        int n_v,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _field_op_batch_mul_sum<<<blocks, threads>>>(res, v, rot, n_v, n);
        return hipGetLastError();
    }

    hipError_t field_op(
        Bn254FrField *res,
        Bn254FrField *l,
        int l_rot,
        Bn254FrField *l_c,
        Bn254FrField *r,
        int r_rot,
        Bn254FrField *r_c,
        int n,
        int op,
        ihipStream_t *stream)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        assert(threads * blocks == n);
        _field_op<<<blocks, threads, 0, stream>>>(res, l, l_rot, l_c, r, r_rot, r_c, n, op);
        return hipGetLastError();
    }

    hipError_t permutation_eval_h_p1(
        Bn254FrField *res,
        const Bn254FrField *first_set,
        const Bn254FrField *last_set,
        const Bn254FrField *l0,
        const Bn254FrField *l_last,
        const Bn254FrField *y,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _permutation_eval_h_p1<<<blocks, threads>>>(res, first_set, last_set, l0, l_last, y, n);
        return hipGetLastError();
    }

    hipError_t permutation_eval_h_p2(
        Bn254FrField *res,
        const Bn254FrField **set,
        const Bn254FrField *l0,
        const Bn254FrField *l_last,
        const Bn254FrField *y,
        int n_set,
        int rot,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _permutation_eval_h_p2<<<blocks, threads>>>(res, set, l0, l_last, y, n_set, rot, n);
        return hipGetLastError();
    }

    hipError_t permutation_eval_h_l(
        Bn254FrField *res,
        const Bn254FrField *beta,
        const Bn254FrField *gamma,
        const Bn254FrField *p,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _permutation_eval_h_l<<<blocks, threads>>>(res, beta, gamma, p, n);
        return hipGetLastError();
    }

    hipError_t ntt(
        Bn254FrField *buf,
        Bn254FrField *tmp,
        const Bn254FrField *pq,
        const Bn254FrField *omegas,
        int log_n,
        int max_deg,
        bool *swap,
        ihipStream_t *stream)
    {
        int p = 0;

        Bn254FrField *src = buf;
        Bn254FrField *dst = tmp;
        int len = 1 << log_n;
        int total = 1 << (log_n - 1);
        while (p < log_n)
        {
            int res = log_n - p;
            int round = (res + max_deg - 1) / max_deg;
            int deg = (res + round - 1) / round;

            int threads = 1 << (deg - 1);
            int blocks = total >> (deg - 1);
            blocks = blocks > 65536 ? 65536 : blocks;
            int grids = (total / blocks) >> (deg - 1);
            _ntt_core<<<blocks, threads, 0, stream>>>(src, dst, pq, omegas, len, p, deg, max_deg, grids);

            Bn254FrField *t = src;
            src = dst;
            dst = t;
            p += deg;
            *swap = !*swap;
        }
        return hipGetLastError();
    }

    hipError_t msm(
        int msm_blocks,
        int max_msm_threads,
        Bn254G1 *res,
        Bn254G1Affine *p,
        Bn254FrField *s,
        int n)
    {
        int threads = n >= max_msm_threads ? max_msm_threads : 1;
        int blocks = (n + threads - 1) / threads;
        _msm_mont_unmont<<<blocks, threads>>>(p, s, false, n);
        _msm_core<<<dim3(32, msm_blocks), threads>>>(res, p, s, n);
        _msm_mont_unmont<<<blocks, threads>>>(p, s, true, n);
        hipDeviceSynchronize();
        return hipGetLastError();
    }

    hipError_t lookup_eval_h(
        Bn254FrField *res,
        const Bn254FrField *input,
        const Bn254FrField *table,
        const Bn254FrField *permuted_input,
        const Bn254FrField *permuted_table,
        const Bn254FrField *z,
        const Bn254FrField *l0,
        const Bn254FrField *l_last,
        const Bn254FrField *l_active_row,
        const Bn254FrField *y,
        const Bn254FrField *beta,
        const Bn254FrField *gamma,
        int rot,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _lookup_eval_h<<<blocks, threads>>>(
            res,
            input, table, permuted_input, permuted_table, z,
            l0, l_last, l_active_row,
            y, beta, gamma,
            rot, n);
        return hipGetLastError();
    }

    hipError_t shuffle_eval_h(
        Bn254FrField *res,
        const Bn254FrField *input,
        const Bn254FrField *table,
        const Bn254FrField *z,
        const Bn254FrField *l0,
        const Bn254FrField *l_last,
        const Bn254FrField *l_active_row,
        const Bn254FrField *y,
        int rot,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _shuffle_eval_h<<<blocks, threads>>>(
            res,
            input, table, z,
            l0, l_last, l_active_row,
            y, rot, n);
        return hipGetLastError();
    }


    hipError_t expand_omega_buffer(
        Bn254FrField *res,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _expand_omega_buffer<<<blocks, threads>>>(res, n);
        return hipGetLastError();
    }

    hipError_t field_mul_zip(
        Bn254FrField *buf,
        Bn254FrField *coeff,
        int coeff_n,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _field_mul_zip<<<blocks, threads>>>(buf, coeff, coeff_n, n);
        return hipGetLastError();
    }

    hipError_t shplonk_h_x_merge(
        Bn254FrField *res,
        Bn254FrField *v,
        Bn254FrField *values,
        Bn254FrField *omegas,
        Bn254FrField *diff_points,
        int diff_points_n,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _shplonk_h_x_merge<<<blocks, threads>>>(res, v, values, omegas, diff_points, diff_points_n, n);
        return hipGetLastError();
    }

    hipError_t shplonk_h_x_div_points(
        Bn254FrField *values,
        Bn254FrField *omegas,
        Bn254FrField *points,
        int points_n,
        int n)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _shplonk_h_x_div_points<<<blocks, threads>>>(values, omegas, points, points_n, n);
        return hipGetLastError();
    }

    hipError_t poly_eval(
        Bn254FrField *p,
        Bn254FrField *res,
        Bn254FrField *tmp,
        const Bn254FrField *x,
        int n)
    {
        Bn254FrField *in = p;
        Bn254FrField *out = res;
        int deg = 0;
        while (n > 1)
        {
            int threads = n / 2 >= 64 ? 64 : 1;
            int blocks = n / threads / 2;
            _poly_eval<<<blocks, threads>>>(in, out, x, deg);
            n >>= 1;

            if (n > 1)
            {
                if (deg == 0)
                {
                    in = res;
                    out = tmp;
                }
                else
                {
                    Bn254FrField *t = in;
                    in = out;
                    out = t;
                }
            }
            deg++;
        }

        if (out != res)
        {
            hipMemcpy(res, out, sizeof(Bn254FrField), hipMemcpyDeviceToDevice);
        }

        return hipGetLastError();
    }

    hipError_t eval_lookup_z(
        Bn254FrField *z,
        Bn254FrField *input,
        Bn254FrField *table,
        Bn254FrField *permuted_input,
        Bn254FrField *permuted_table,
        Bn254FrField *beta_gamma,
        int n,
        ihipStream_t *stream)
    {
        int threads = n >= 64 ? 64 : 1;
        int blocks = n / threads;
        _eval_lookup_z_step1<<<blocks, threads, 0, stream>>>(
            z, permuted_input, permuted_table, beta_gamma);

        int worker = 64 * 128;
        int size_per_worker = n / worker;
        _eval_lookup_z_batch_invert<<<128, 64, 0, stream>>>(
            z, permuted_input, size_per_worker);

        _eval_lookup_z_step2<<<blocks, threads, 0, stream>>>(
            z, input, table, beta_gamma);

        worker = 64 * 64;
        size_per_worker = n / worker;
        _eval_lookup_z_product_batch<<<64, 64, 0, stream>>>(
            z, permuted_input, size_per_worker);
        _eval_lookup_z_product_batch<<<8, 8, 0, stream>>>(
            permuted_input, permuted_table, 64);
        _eval_lookup_z_product_single_spread<<<1, 1, 0, stream>>>(
            permuted_table, 64);
        _eval_lookup_z_product_batch_spread<<<8, 8, 0, stream>>>(
            permuted_input, permuted_table, 64);
        _eval_lookup_z_product_batch_spread_skip<<<64, 64, 0, stream>>>(
            z, permuted_input, size_per_worker);

        return hipGetLastError();
    }
}
